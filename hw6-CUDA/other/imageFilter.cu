
#include <hip/hip_runtime.h>
#include <stdio.h>

// A macro for checking the error codes of cuda runtime calls
#define CUDA_ERROR_CHECK(expr) \
  {                            \
    hipError_t err = expr;    \
    if (err != hipSuccess)    \
    {                          \
      printf("CUDA call failed!\n%s\n", hipGetErrorString(err)); \
      exit(1);                 \
    }                          \
  }


__global__
void swapChannel_kernel(uchar3 * device_inputImage, uchar3 * device_outputImage, int rows, int cols)
{
  //----------------------------------------------------------------
  // TODO: Implement this Kernel
  //----------------------------------------------------------------
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  device_outputImage[idx].y = device_inputImage[idx].x;
  device_outputImage[idx].x = device_inputImage[idx].y;
  device_outputImage[idx].z = device_inputImage[idx].z;

}

__global__
void blurImage_kernel(uchar3 * device_inputImage, uchar3 * device_outputImage, int rows, int cols)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  uchar3 color;
  color.x = color.y = color.z =0;

  int margin_up = blockIdx.x; 
  int margin_down = 511 - blockIdx.x;
  int margin_right = 511 - threadIdx.x;
  int margin_left = threadIdx.x;
  int mini = margin_up;


  if(margin_left >= 4 && margin_right >= 4 && margin_up >= 4 && margin_down >= 4 ){

          for(int i = -4; i < 4 ; i++){
            for(int j = -4; j < 4 ; j++){
            color.x += device_inputImage[(blockIdx.x+i)*215+(threadIdx.x+j)].x;
            color.y += device_inputImage[(blockIdx.x+i)*215+(threadIdx.x+j)].y;
            color.z += device_inputImage[(blockIdx.x+i)*215+(threadIdx.x+j)].z;
            }
          }
        color.x /= 9;
        color.y /= 9;
        color.z /= 9;
  }

  else{

  if(margin_down < mini)
    mini = margin_down;

  if(margin_right < mini)
    mini = margin_right;

  if(margin_left < mini)
    mini = margin_left;

  for(int i = (0-mini); i < mini ; i++){
            for(int j = (0-mini); j < mini ; j++){
            color.x += device_inputImage[(blockIdx.x+i)*215+(threadIdx.x+j)].x;
            color.y += device_inputImage[(blockIdx.x+i)*215+(threadIdx.x+j)].y;
            color.z += device_inputImage[(blockIdx.x+i)*215+(threadIdx.x+j)].z;
            }
          }
        color.x /= (2*mini+1);
        color.y /= (2*mini+1);
        color.z /= (2*mini+1);
  }
  device_outputImage[idx] = color;
}

__global__
void inplaceFlip_kernel(uchar3 * device_outputImage, int rows, int cols)
{
  //----------------------------------------------------------------
  // TODO: Implement this Kernel
  //----------------------------------------------------------------

}

__global__
void creative_kernel(uchar3 * device_inputImage, uchar3 * device_outputImage, int rows, int cols)
{
  //----------------------------------------------------------------
  // TODO: Implement this Kernel
  //----------------------------------------------------------------

}


__host__
float filterImage(uchar3 *host_inputImage, uchar3 *host_outputImage, int rows, int cols, int filterNumber){

  int numPixels = rows * cols;

  //allocate memory on device (GPU)
  uchar3 *device_inputImage;
  uchar3 *device_outputImage;

  CUDA_ERROR_CHECK(hipMalloc(&device_inputImage, sizeof(uchar3) * numPixels));
  CUDA_ERROR_CHECK(hipMalloc(&device_outputImage, sizeof(uchar3) * numPixels));
  CUDA_ERROR_CHECK(hipMemset(device_outputImage, 0,  sizeof(uchar3) * numPixels)); //make sure no memory is left laying around

  //copy input image to the device (GPU)
  CUDA_ERROR_CHECK(hipMemcpy(device_inputImage, host_inputImage, sizeof(uchar3) * numPixels, hipMemcpyHostToDevice));

  //start timing to measure length of kernel call
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  //----------------------------------------------------------------
  // TODO: Fill in the parameters for the kernel calls
  //----------------------------------------------------------------
  // Each of the parameters are as follows:
  //    1. Number of thread blocks, can be either int or dim3 (see CUDA manual)
  //    2. Number of threads per thread block, can be either int or dim3 (see CUDA manual)

  // Also note that you pass the pointers to the device memory to the kernel call

  int gridSize = 512;
  int blockSize = 512;

  switch(filterNumber){
    case 1:
      swapChannel_kernel<<<gridSize,blockSize>>>(device_inputImage, device_outputImage, rows, cols);
      break;
    case 2:
      blurImage_kernel<<<gridSize,blockSize>>>(device_inputImage, device_outputImage, rows, cols);
      break;
    case 3:
      inplaceFlip_kernel<<<gridSize,blockSize>>>(device_inputImage, rows, cols);
      break;
    case 4:
      creative_kernel<<<gridSize,blockSize>>>(device_inputImage, device_outputImage, rows, cols);
      break;
    default:
      break;
  }

  //----------------------------------------------------------------
  // END KERNEL CALLS - Do not modify code beyond this point!
  //----------------------------------------------------------------

  //stop timing
  hipEventRecord(stop);

  hipEventSynchronize(stop);

  float timeElapsedInMs = 0;
  hipEventElapsedTime(&timeElapsedInMs, start, stop);

  //synchronize
  hipDeviceSynchronize(); CUDA_ERROR_CHECK(hipGetLastError());

  //copy device output image back to host output image
  //special case for filter swap - since it is in place, we actually copy the input image back to the host output
  if (filterNumber==3){
    CUDA_ERROR_CHECK(hipMemcpy(host_outputImage, device_inputImage, sizeof(uchar3) * numPixels, hipMemcpyDeviceToHost));
  }else{
    CUDA_ERROR_CHECK(hipMemcpy(host_outputImage, device_outputImage, sizeof(uchar3) * numPixels, hipMemcpyDeviceToHost));
  }


  //free Memory
  CUDA_ERROR_CHECK(hipFree(device_inputImage));
  CUDA_ERROR_CHECK(hipFree(device_outputImage));

  return timeElapsedInMs;
}
